#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "timer.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void kernel(const int* in, int* out, int N, int radius)
{
    extern __shared__ int shared[]; // dim: blocksize + 2 radius
    int* tile = shared;

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= N)
        return;

    int tdx = threadIdx.x + radius; // current thread index in tile

    tile[tdx] = in[idx]; // central element (each thread)

    // leftmost threads load halo elements
    if (threadIdx.x < radius)
    {
        tile[threadIdx.x] = (idx < radius) ? 0 /* global leftmost oob (first block) */ : in[idx - radius];                   /* left stencil */
        tile[tdx + blockDim.x] = (idx + blockDim.x >= N) /* global rightmost oob (last block) */ ? 0 : in[idx + blockDim.x]; /* right stencil */
    }

    __syncthreads();

    if (idx >= radius && idx < N - radius)
    {
        int result = 0;
        for (int offset = -radius; offset <= radius; offset++)
        {
            result += tile[tdx + offset];
        }
        out[idx] = result;
    }
    else
    {
        out[idx] = tile[threadIdx.x]; // pad block position
    }
}

namespace cuda
{
    void stencil_1d(const int* in, int* out, int N, int radius, int blockSize)
    {
        int *d_in, *d_out;

        int size = N * sizeof(int);

        hipMalloc(&d_in, size);
        hipMalloc(&d_out, size);

        {
            ScopedTimer t1("memcpy inputs CPU -> GPU", POST);
            hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
            hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);
        }

        {
            ScopedTimer t2("kernel execution", POST);
            kernel<<<(N + blockSize - 1) / blockSize, blockSize, (blockSize + 2 * radius) * sizeof(int)>>>(d_in, d_out, N, radius);
            CUDA_CHECK
            hipDeviceSynchronize();
            CUDA_CHECK
        }

        {
            ScopedTimer t3("memcpy output GPU -> CPU", POST);
            hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
        }

        /*
        // Error Checking
        for (int i = radius; i < N; i++)
        {
            if (i < radius || i >= N - radius)
            {
                // Halo elements should remain as 1
                if (out[i] != 1)
                    printf("Mismatch at index %d, was: %d, should be: %d\n", i, out[i], 1);
            }
            else
            {
                // Stencil sum should be 1 + 2 * RADIUS (since input is initialized to 1)
                if (out[i] != 1 + 2 * radius)
                    printf("Mismatch at index %d, was: %d, should be: %d\n", i, out[i], 1 + 2 * radius);
            }
        }
        */

        hipFree(d_in);
        hipFree(d_out);
    }
}